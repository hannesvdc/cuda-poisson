#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>
#include <algorithm>
#include <chrono>

// Index helper function
__device__
int getArrayIndex(int x_index, int y_index, int N) {
    return N * y_index + x_index;
}

__global__
void initializeRHS(float *f, float dx, int N) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // global x-index = block_size * which block am I + which thread am I?
    int y = blockIdx.y * blockDim.y + threadIdx.y; // global x-index = block_size * which block am I + which thread am I?

    float x_coord = x * dx;
    float y_coord = y * dx;
    int index = getArrayIndex(x, y, N);
    f[index] = -sinf(2.0 * M_PI * x_coord) * sinf(M_PI * y_coord);
}

__global__
void jacobi_step(float *u_old, float *u_new, float *f, float dx, int N) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // global x-index = block_size * which block am I + which thread am I?
    int y = blockIdx.y * blockDim.y + threadIdx.y; // global x-index = block_size * which block am I + which thread am I?

    if ( x > 0 && x < N-1 && y > 0 && y < N-1 ) {
        int index = getArrayIndex( x, y, N);
        int left  = getArrayIndex( x-1, y, N);
        int right = getArrayIndex( x+1, y, N);
        int up    = getArrayIndex( x, y+1, N);
        int down  = getArrayIndex( x, y-1, N);

        // Update into new array with Jacobi
        u_new[index] = 0.25f * (u_old[left] + u_old[right] + u_old[up] + u_old[down] - dx * dx * f[index]);
    }
}

void writeSolutionToCSV(const char* filename, float* u, int N) {
    std::ofstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Error opening file for writing.\n";
        return;
    }

    for (int y = 0; y < N; ++y) {
        for (int x = 0; x < N; ++x) {
            int index = y * N + x;
            file << u[index];
            if (x != N - 1) file << ",";
        }
        file << "\n";
    }

    file.close();
}

int main() {
    // Allocate u_old, u_new and f in regular memory.
    int N = 256;
    int n_elements = N * N;
    size_t size_bytes = n_elements * sizeof(float);
    float *u_old = new float[n_elements]();
    float dx = 1.0f / (N-1);

    // Device memory
    float *u_old_d, *u_new_d, *f_d;
    hipMalloc(&u_old_d, size_bytes);
    hipMalloc(&u_new_d, size_bytes);
    hipMalloc(&f_d, size_bytes);

    // Define the Grid and Block sizes for the number of threads
    int nThreadsPerDim = 16;
    int nBlocksPerDim = 16;
    dim3 blockDim(nThreadsPerDim, nThreadsPerDim);
    dim3 gridDim(nBlocksPerDim, nBlocksPerDim);

    // Copy source term f and initial u_old to device
    hipMemcpy(u_old_d, u_old, size_bytes, hipMemcpyHostToDevice);
    initializeRHS<<<gridDim, blockDim>>>(f_d, dx, N);
    std::cout << "Launching grid of size (" << gridDim.x << ", " << gridDim.y << ") "
              << "with blocks of size (" << blockDim.x << ", " << blockDim.y << ")\n";


    // Call the Jacobi solver iteratively
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    int max_iter = 100000;
    for (int iter = 0; iter < max_iter; ++iter) {
        jacobi_step<<<gridDim, blockDim>>>(u_old_d, u_new_d, f_d, dx, N);

        // Swap pointers (u_old becomes input for next iteration)
        std::swap(u_old_d, u_new_d);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU time: " << milliseconds / 1000.0f << " seconds" << std::endl;

    // Move the solution u_old_d to the CPU
    hipDeviceSynchronize();
    hipMemcpy(u_old, u_old_d, size_bytes, hipMemcpyDeviceToHost);

    // Store result in CSV file
    writeSolutionToCSV("solution_cuda.csv", u_old, N);

    // Free memory
    hipFree(u_old_d);
    hipFree(u_new_d);
    hipFree(f_d);
    delete[] u_old;

    return 0;
}